
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void MatrixMulKernel(float* M, float* N, float* P, int Width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if ((row < Width) && (col < Width)) {
        float Pvalue = 0;
        for (int k = 0; k < Width; ++k) {
            Pvalue += M[row * Width + k] * N[k * Width + col];
        }
        P[row * Width + col] = Pvalue;
    }
}

int main(int argc, char* argv[]) {
    if (argc != 3) {
        printf("Usage: %s <Matrix Width> <Block Size>\n", argv[0]);
        return 1;
    }

    int Width = atoi(argv[1]);
    int BlockSize = atoi(argv[2]);

    int size = Width * Width * sizeof(float);

    // Allocate host memory
    float *h_M = (float*)malloc(size);
    float *h_N = (float*)malloc(size);
    float *h_P = (float*)malloc(size);

    // Initialize input matrices
    for (int i = 0; i < Width * Width; ++i) {
        h_M[i] = 1.0f;  // or random values
        h_N[i] = 1.0f;
    }

    // Allocate device memory
    float *d_M, *d_N, *d_P;
    hipMalloc((void**)&d_M, size);
    hipMalloc((void**)&d_N, size);
    hipMalloc((void**)&d_P, size);

    hipMemcpy(d_M, h_M, size, hipMemcpyHostToDevice);
    hipMemcpy(d_N, h_N, size, hipMemcpyHostToDevice);

    // Setup execution configuration
    dim3 dimBlock(BlockSize, BlockSize);
    int blocksPerGridX = (Width + BlockSize - 1) / BlockSize;
    int blocksPerGridY = (Width + BlockSize - 1) / BlockSize;
    dim3 dimGrid(blocksPerGridX, blocksPerGridY);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    // Launch kernel
    MatrixMulKernel<<<dimGrid, dimBlock>>>(d_M, d_N, d_P, Width);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipDeviceSynchronize();

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("KERNEL_TIME: %.3f ms\n", milliseconds);

    // Immediately check for launch errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
    }

    // Optional but useful for debugging and correctness
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(h_P, d_P, size, hipMemcpyDeviceToHost);

    // Print first few results (optional)
    printf("Sample output:\n");
    for (int i = 0; i < (Width > 4 ? 4 : Width); ++i) {
        for (int j = 0; j < (Width > 4 ? 4 : Width); ++j) {
            printf("%0.1f ", h_P[i * Width + j]);
        }
        printf("\n");
    }

    // Cleanup
    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P);
    free(h_M);
    free(h_N);
    free(h_P);

    return 0;
}
