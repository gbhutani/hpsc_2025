#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <windows.h>

#define N 10240000

__global__ void vector_add(float *a, float *b, float *c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
        c[idx] = a[idx] + b[idx];
}

int main() {
    int size = N * sizeof(float);
    float *h_a, *h_b, *h_c;
    float *d_a, *d_b, *d_c;

    h_a = (float *)malloc(size);
    h_b = (float *)malloc(size);
    h_c = (float *)malloc(size);

    for (int i = 0; i < N; i++) {
        h_a[i] = i * 1.0f;
        h_b[i] = (N - i) * 1.0f;
    }

    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vector_add<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("CUDA error: %s\n", hipGetErrorString(err));

    Sleep(5000);  // to see GPU memory usage in Task Manager

    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < 5; i++) {
        printf("%d: %f + %f = %f\n", i, h_a[i], h_b[i], h_c[i]);
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
